#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <cstdio>

//const int block_size = 16;
/*
   0-5 color +-x, +-y, +-z;
   6 normals
   7 father
   8 child
   9 info
   last byte
1: is procesing
2: is leaves
4: is first son

10 light
 */
const int block_size = 11;
const int idx_normal = 6;
const int idx_father = 7;
const int idx_child = 8;
const int idx_info = 9;
const int memory_size = 2000000;

const int leave_size = 3;

/*
leaves:
r g b a
x y z a
c: x y z

queue:
leavesidx, treeidx, level,
 */

__device__ int
cudaCombine(int a, int b, int c, int d) {
    return ((a & 0xFF) << 24) + ((b & 0xFF) << 16)
        + ((c & 0xFF) << 8) + (d & 0xFF);
}

/*****************************************************************************/
/**Find all valid leaves******************************************************/
/*****************************************************************************/
__global__ void
cudaFindLeavesThreads(int* colors, int* normals, int* leaves, int* p, int dim) {
    int x = threadIdx.x;
    int y = blockIdx.x % blockDim.x;
    int z = blockIdx.x / blockDim.x;

    int element;
    element = colors[x * dim * dim + y * dim + z];
    //     	surf3Dread(&element, colors, x * sizeof(uint32_t), y, z);
    if (element & 0xFF) {
        int idx = atomicAdd(p, 1) * leave_size;
        leaves[idx] = element;
        leaves[idx + 1] = normals[x * dim * dim + y * dim + z];
        leaves[idx + 2] = cudaCombine(x, y, z, 0);
    }
}

int cudaFindLeaves(int* colors, int* normals, int* leaves, int* p, int dim) {
    hipMemset(p, 0, sizeof(int));
    int threadsPerBlock = dim;
    int blocks = dim * dim;
    cudaFindLeavesThreads<<<blocks, threadsPerBlock>>>(colors, normals, leaves, p, dim);

    int num_leaves;
    hipMemcpy(&num_leaves, p, sizeof(int), hipMemcpyDeviceToHost);
    return num_leaves;
}

/*****************************************************************************/
/**build tree details*********************************************************/
/*****************************************************************************/
__device__ void
cudaSetLeave(int tree_idx, int* tree, int* leave) {
    for (int i = 0; i < 6; ++i) {
        tree[i] = leave[0];
    }
    tree[idx_normal] = leave[1];
    leave[2] = tree_idx;
}

__device__ void
cudaAddToTree(int leave_idx, int* leaves,
        int tree_idx, int* tree, int level, int* ptrTree,
        int* queue, int* ptrQueue) {

    int* tree_element = &tree[tree_idx * block_size];

    // is leaves
    if (level == 1) {
        cudaSetLeave(tree_idx, tree_element, &leaves[leave_idx * leave_size]);
        tree_element[idx_info] += 2;
        return ;
    }
    // has child finish processing
    if (tree_element[idx_child] && (tree_element[idx_info] & 1) == 0) {		
        int* leave_element = &leaves[leave_idx * leave_size];
        int new_level = level >> 1;
        bool x = ((leave_element[2] & 0xFF000000) >> 24) & new_level;
        bool y = ((leave_element[2] & 0x00FF0000) >> 16) & new_level;
        bool z = ((leave_element[2] & 0x0000FF00) >>  8) & new_level;
        int new_idx = tree_element[idx_child] + x * 4 + y * 2 + z;
        cudaAddToTree(leave_idx, leaves, new_idx, tree, new_level, ptrTree, queue, ptrQueue);
    } else {
        // try to process
        bool processing = atomicOr(&tree_element[idx_info], 1) & 1;
        if (processing) {
            int ptr = atomicAdd(ptrQueue, 1);
            queue[ptr * 3] = leave_idx;
            queue[ptr * 3 + 1] = tree_idx;
            queue[ptr * 3 + 2] = level;
        } else {
            // process
            int idx = atomicAdd(ptrTree, 8) + 1;
            tree_element[idx_child] = idx;
            for (int i = 0; i < 8; ++i) {
                tree[(idx + i) * block_size + idx_father] = tree_idx;
            }
            tree[idx * block_size + idx_info] += 4;

            // finish processing
            atomicAnd(&tree_element[idx_info], 0xFFFFFFFE);

            int* leave_element = &leaves[leave_idx * leave_size];
            int new_level = level >> 1;
            bool x = ((leave_element[2] & 0xFF000000) >> 24) & new_level;
            bool y = ((leave_element[2] & 0x00FF0000) >> 16) & new_level;
            bool z = ((leave_element[2] & 0x0000FF00) >>  8) & new_level;
            int new_idx = tree_element[idx_child] + x * 4 + y * 2 + z;
            cudaAddToTree(leave_idx, leaves, new_idx, tree, new_level, ptrTree, queue, ptrQueue);
        }
    }
}

__global__ void
cudaBuildTreeFromLeaves(int* leaves, int num_leaves, int* tree, int* ptrTree, int* queue, int* ptrQueue, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_leaves) {
        cudaAddToTree(idx, leaves, 0, tree, dim, ptrTree, queue, ptrQueue);
    }
}

__global__ void
cudaBuildTreeFromQueue(int* leaves, int num_leaves, int* tree, int* ptrTree, int* queue, int* ptrQueue, int rangeL, int rangeR, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + rangeL;
    if (idx < rangeR) {
        int* queue_element = &queue[idx * leave_size];
        cudaAddToTree(queue_element[0], leaves, queue_element[1], tree, queue_element[2], ptrTree, queue, ptrQueue);
    }
}

void cudaBuildTree(int* leaves, int num_leaves, int* queue, int* tree, int dim, int* ptrTree, int* ptrQueue) {
    hipMemset(ptrTree, 0, sizeof(int));
    hipMemset(ptrQueue, 0, sizeof(int));

    int fr, la, total = dim * dim * dim;
    int threadsPerBlock = dim, blocks = (num_leaves - 1 + threadsPerBlock) / threadsPerBlock;
    cudaBuildTreeFromLeaves<<<blocks, threadsPerBlock>>>(leaves, num_leaves, tree, ptrTree, queue, ptrQueue, dim);

    fr = 0;
    hipMemcpy(&la, ptrQueue, sizeof(int), hipMemcpyDeviceToHost);
    while (la > fr) {
        if (la >= total) {
            hipMemset(ptrQueue, 0, sizeof(int));
        }

        blocks = (la - fr - 1 + threadsPerBlock) / threadsPerBlock;
        cudaBuildTreeFromQueue<<<blocks, threadsPerBlock>>>(leaves, num_leaves, tree, ptrTree, queue, ptrQueue, fr, la, dim);

        fr = la >= total ? 0:la;
        hipMemcpy(&la, ptrQueue, sizeof(int), hipMemcpyDeviceToHost);
    }
}

/*****************************************************************************/
/***mix up info***************************************************************/
/*****************************************************************************/
__global__ void
cudaCombineFromLeavesThread(int* tree, int* leaves, int num_leaves, int* queue, int* ptrQueue) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_leaves) {
        int tree_idx = leaves[idx * leave_size + 2];
        int* element = &tree[tree_idx * block_size];
        if (element[idx_info] & 4) {
            int new_idx = atomicAdd(ptrQueue, 1);
            queue[new_idx] = element[idx_father];
        }
    }
}

__device__ void
addColor(int* c0, int* c1) {
    c0[0] += ((*c1) & 0xFF000000) >> 24;
    c0[1] += ((*c1) & 0x00FF0000) >> 16;
    c0[2] += ((*c1) & 0x0000FF00) >> 8; 
    c0[3] += (*c1) & 0xFF;
}

__device__ int
alphaBlend(int* c0, int* c1) {
    int a0 = c0[3], a1 = c1[3], a_new = 255 - a0;

    int r = int((c0[0] * a0 + c1[0] * a_new) / 4 / 255.0f) & 0xFF;
    int g = int((c0[1] * a0 + c1[1] * a_new) / 4 / 255.0f) & 0xFF;
    int b = int((c0[2] * a0 + c1[2] * a_new) / 4 / 255.0f) & 0xFF;
    int a = int((c0[0] * a0 + a1 * a_new) / 4) & 0xFF;

    return cudaCombine(r, g, b, a);
}

__global__ void
cudaCombineFromQueueThread(int* tree, int rangeL, int rangeR, int* queue, int* ptrQueue) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + rangeL;
    if (idx < rangeR) {
        int* element = &tree[queue[idx] * block_size];

        // color
        int xx[16], yy[16], zz[16];
        memset(xx, 0, sizeof(xx));
        memset(yy, 0, sizeof(yy));
        memset(zz, 0, sizeof(zz));

        for (int x = 0; x < 2; ++x) {
            for (int y = 0; y < 2; ++y) {
                for (int z = 0; z < 2; ++z) {
                    int idx_i = x * 4 + y * 2 + z;

                    if (x) {
                        addColor(&xx[0], &tree[(element[idx_child] + idx_i) * block_size]);
                        addColor(&xx[12], &tree[(element[idx_child] + idx_i) * block_size + 1]);
                    } else {
                        addColor(&xx[4], &tree[(element[idx_child] + idx_i) * block_size]);
                        addColor(&xx[8], &tree[(element[idx_child] + idx_i) * block_size + 1]);
                    }

                    if (y) {
                        addColor(&yy[0], &tree[(element[idx_child] + idx_i) * block_size + 2]);
                        addColor(&yy[12], &tree[(element[idx_child] + idx_i) * block_size + 3]);
                    } else {
                        addColor(&yy[4], &tree[(element[idx_child] + idx_i) * block_size + 2]);
                        addColor(&yy[8], &tree[(element[idx_child] + idx_i) * block_size + 3]);
                    }

                    if (z) {
                        addColor(&zz[0], &tree[(element[idx_child] + idx_i) * block_size + 4]);
                        addColor(&zz[12], &tree[(element[idx_child] + idx_i) * block_size + 5]);
                    } else {
                        addColor(&zz[4], &tree[(element[idx_child] + idx_i) * block_size + 4]);
                        addColor(&zz[8], &tree[(element[idx_child] + idx_i) * block_size + 5]);
                    }
                }
            }
        }

        element[0] = alphaBlend(&xx[0], &xx[4]);
        element[1] = alphaBlend(&xx[8], &xx[12]);
        element[2] = alphaBlend(&yy[0], &yy[4]);
        element[3] = alphaBlend(&yy[8], &yy[12]);
        element[4] = alphaBlend(&zz[0], &zz[4]);
        element[5] = alphaBlend(&zz[8], &zz[12]);

        //		element[6] = normalMix();

        if (element[idx_info] & 4) {
            int new_idx = atomicAdd(ptrQueue, 1);
            queue[new_idx] = element[idx_father];
        }
    }
}

void cudaCombine(int* tree, int* leaves, int num_leaves, int* queue, int* ptrQueue) {
    int threadsPerBlock = 256;
    int blocks = (num_leaves - 1 + threadsPerBlock) / threadsPerBlock;
    hipMemset(ptrQueue, 0, sizeof(int));

    int st = 0, en;
    cudaCombineFromLeavesThread<<<blocks, threadsPerBlock>>>(tree, leaves, num_leaves, queue, ptrQueue);

    hipMemcpy(&en, ptrQueue, sizeof(int), hipMemcpyDeviceToHost);
    while (en > st) {
        blocks = (en - st - 1 + threadsPerBlock) / threadsPerBlock;
        cudaCombineFromQueueThread<<<blocks, threadsPerBlock>>>(tree, st, en, queue, ptrQueue);

        st = en;
        hipMemcpy(&en, ptrQueue, sizeof(int), hipMemcpyDeviceToHost);
    }
}

/*****************************************************************************/
/***wrap up*******************************************************************/
/*****************************************************************************/
int cudaBuildTreeOverall(int* colors, int* normals, int* leaves, int* tree, int* queue, int dim) {
    int *ptr0, *ptr1;
    hipMalloc(&ptr0, sizeof(int));
    hipMalloc(&ptr1, sizeof(int));
    hipMemset(tree, 0, sizeof(int) * memory_size * block_size);

    int num_leaves = cudaFindLeaves(colors, normals, leaves, ptr0, dim);
    cudaBuildTree(leaves, num_leaves, queue, tree, dim, ptr0, ptr1);

    cudaCombine(tree, leaves, num_leaves, queue, ptr0);

    hipFree(ptr0);
    hipFree(ptr1);

    return num_leaves;
}

// build tree for test
void cudaBuildMalloc(int* colors, int* normals, int voxel_dim, int* res) {
    int *gpu_colors, *gpu_normals, *gpu_tmp, *gpu_res, *gpu_queue;

    int total = voxel_dim * voxel_dim * voxel_dim;
    hipMalloc(&gpu_colors, sizeof(int) * total);
    hipMemcpy(gpu_colors, colors, sizeof(int) * total, hipMemcpyHostToDevice);
    hipMalloc(&gpu_normals, sizeof(int) * total);
    hipMemcpy(gpu_normals, normals, sizeof(int) * total, hipMemcpyHostToDevice);
    hipMalloc(&gpu_tmp, sizeof(int) * memory_size * 6);
    hipMalloc(&gpu_res, sizeof(int) * memory_size * block_size);
    hipMalloc(&gpu_queue, sizeof(int) * total);

    int len = cudaBuildTreeOverall(gpu_colors, gpu_normals, gpu_tmp, gpu_res, gpu_queue, voxel_dim);
    printf("len = %d\n", len);
}

/*****************************************************************************/

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
                static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
